
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "kernel/kernel_compat_cuda.h"
#include "kernel_config.h"

#include "util/util_atomic.h"

#include "kernel/kernel_math.h"
#include "kernel/kernel_types.h"
#include "kernel/kernel_globals.h"
#include "kernel/kernel_color.h"
#include "kernel/kernels/cuda/kernel_cuda_image.h"
#include "kernel/kernel_film.h"
#include "kernel/kernel_path.h"
#include "kernel/kernel_path_branched.h"
#include "kernel/kernel_bake.h"
#include "kernel/kernel_work_stealing.h"

/* kernels */
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);
	bool thread_is_active = work_index < total_work_size;
	uint x, y, sample;
	KernelGlobals kg;
	if(thread_is_active) {
		get_work_pixel(tile, work_index, &x, &y, &sample);

		kernel_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}

	if(kernel_data.film.cryptomatte_passes) {
		__syncthreads();
		if(thread_is_active) {
			kernel_cryptomatte_post(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
		}
	}
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(WorkTile *tile, uint total_work_size)
{
	int work_index = ccl_global_id(0);
	bool thread_is_active = work_index < total_work_size;
	uint x, y, sample;
	KernelGlobals kg;
	if(thread_is_active) {
		get_work_pixel(tile, work_index, &x, &y, &sample);

		kernel_branched_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}

	if(kernel_data.film.cryptomatte_passes) {
		__syncthreads();
		if(thread_is_active) {
			kernel_cryptomatte_post(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
		}
	}
}
#endif

#if 0
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int height, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, height, offset, stride);
	}
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_float(float *rgba, float *buffer, float sample_scale, int pass_type, int sx, int sy, int sw, int sh, int height, int offset, int stride, int fullw, int fullh, int pixelsize)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh) {
		kernel_film_convert_to_float(NULL, rgba, buffer, sample_scale, pass_type, x, y, height, offset, stride, fullw, fullh, pixelsize);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_displace(uint4 *input,
                     float4 *output,
                     int type,
                     int sx,
                     int sw,
                     int offset,
                     int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_displace_evaluate(&kg, input, output, x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_background(uint4 *input,
                       float4 *output,
                       int type,
                       int sx,
                       int sw,
                       int offset,
                       int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_background_evaluate(&kg, input, output, x);
	}
}

#ifdef __BAKING__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(uint4 *input, float4 *output, int type, int filter, int sx, int sw, int offset, int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		KernelGlobals kg;
		kernel_bake_evaluate(&kg, input, output, (ShaderEvalType)type, filter, x, offset, sample);
	}
}
#endif

#endif

